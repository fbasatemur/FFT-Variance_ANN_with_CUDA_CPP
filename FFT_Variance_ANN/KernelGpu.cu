#include "hip/hip_runtime.h"

#include ""
#include "CpuGpuMat.h"
#include "KernelGpu.cuh"
#include <math.h>
// #include <stdio.h>

/*
	 __global__ prefix'i vscc tarafindan anlasilmaz ve bu fonksiyonlari nvcc compile edecektir
	 nvcc'in compile edecegi metodlar "KernelGpu.cuh" icerisinde declare edilir ve burada tanimlanmalidir
 */

__global__ void gpuMatrixMult(float* gpuMat1, float* gpuMat2, float* gpuMat3, int m1Rows, int m1Cols, int m2Cols)
{
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	float sum = 0.0;

	// printf("BIY:%d BDY:%d TIY:%d BIX:%d BDX:%d TIX:%d r:%d c:%d \n", blockIdx.y, blockDim.y, threadIdx.y, blockIdx.x, blockDim.x, threadIdx.x, row, col);

	if (row < m1Rows && col < m2Cols) {
		for (int i = 0; i < m1Cols; i++) {

			sum += gpuMat1[row * m1Cols + i] * gpuMat2[i * m2Cols + col];
		}
		/*printf("%f\n", sum);*/
		gpuMat3[row * m2Cols + col] = sum;
	}
}

__global__ void gpuNormAndShift(float* GpuP, int size, float mean, float variance, float epsilon, float beta, float gamma)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;

	if (id < size) {
		GpuP[id] = (GpuP[id] - mean) / sqrt(variance + epsilon) * gamma + beta;
	}
}

__global__ void gpuReluActivation(float* GpuP, int size)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;

	if (id < size) {
		GpuP[id] = GpuP[id] > 0 ? GpuP[id] : 0;
	}
}
__global__ void gpuSigmoidActivation(float* GpuP, int size)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;

	if (id < size) {
		GpuP[id] = (float)(1.0 / (1.0 + exp(-1.0 * (double)GpuP[id])));
	}
}

__global__ void gpuBatchNorm(float* gpuResult, float* gpuBeta, float* gpuGamma, float* gpuMovingMean, float* gpuMovingVar, float epsilon, int size)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;

	if (id < size) {
		gpuResult[id] = (gpuResult[id] - gpuMovingMean[id]) / sqrt(gpuMovingVar[id] + epsilon) * gpuGamma[id] + gpuBeta[id];
	}
}


/*
	"gpuMatrixMult" fonksiyonunu yalnizca .cu uzantili dosyada cagirabiliriz.
	Sebebi bu fonksiyonlari vscc degil de nvcc in compile etmesini saglamak icindir.
	vscc gpuMatrixMult'e kadar compile eder; gpuMatrixMult fonksiyonunu ise nvcc compile eder ve paralel kosar.
*/

void gpuMatrixMultiplication(CpuGpuMat* Mat1, CpuGpuMat* Mat2, CpuGpuMat* Mat3)
{
	//vscc
	int threadsPerBlock = 32;

	int gridCols = ceil(double(Mat2->Cols) / double(threadsPerBlock));
	int gridRows = ceil(double(Mat1->Rows) / double(threadsPerBlock));

	dim3 gridDim(gridCols, gridRows);
	dim3 blockDim(threadsPerBlock, threadsPerBlock);

	//nvcc
	gpuMatrixMult << < gridDim, blockDim >> > ((float*)Mat1->GpuP, (float*)Mat2->GpuP, (float*)Mat3->GpuP, Mat1->Rows, Mat1->Cols, Mat2->Cols);
}


void gpuNormalizeAndShift(CpuGpuMat* Mat, float mean, float variance, float epsilon, float beta, float gama)
{
	int threadsPerBlock = 32;
	int blocksPerGrid = ceil(double(Mat->Size) / double(threadsPerBlock));

	gpuNormAndShift << < blocksPerGrid, threadsPerBlock >> > ((float*)Mat->GpuP, Mat->Size, mean, variance, epsilon, beta, gama);
}

void gpuRelu(CpuGpuMat* Mat)
{
	int threadsPerBlock = 32;
	int blocksPerGrid = ceil(double(Mat->Size) / double(threadsPerBlock));

	gpuReluActivation << < blocksPerGrid, threadsPerBlock >> > ((float*)Mat->GpuP, Mat->Size);
}

void gpuSigmoid(CpuGpuMat* Mat) {

	int threadsPerBlock = 32;
	int blocksPerGrid = ceil(double(Mat->Size) / double(threadsPerBlock));

	gpuSigmoidActivation << < blocksPerGrid, threadsPerBlock >> > ((float*)Mat->GpuP, Mat->Size);
}

void gpuBatchNormalization(CpuGpuMat* result, CpuGpuMat* beta, CpuGpuMat* gamma, CpuGpuMat* movingMean, CpuGpuMat* movingVariance, float epsilon) {

	int threadsPerBlock = 32;
	int blocksPerGrid = ceil(double(beta->Size) / double(threadsPerBlock));

	gpuBatchNorm << < blocksPerGrid, threadsPerBlock >> > ((float*)result->GpuP, (float*)beta->GpuP, (float*)gamma->GpuP, (float*)movingMean->GpuP, (float*)movingVariance->GpuP, epsilon, beta->Size);
}